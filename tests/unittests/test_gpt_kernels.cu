#include <vector>
#include <random>

#include "kernels/gpt_kernels.h"
#include "utils/memory_utils.h"
#include "gtest_utils.h"

using namespace space_llm;

int test_find_context_dups();
int test_compact();

int main(int argc, char *argv[]) {
    bool all_passed = true;
    bool passed;

    // passed = test_find_context_dups() == EXIT_SUCCESS;
    // all_passed |= passed;
    // printf("%s", passed ? "." : "X");
    // if (!passed) {
    //     puts("\ntest_find_context_dups: FAILED");
    // }

    passed = test_compact() == EXIT_SUCCESS;
    all_passed |= passed;
    printf("%s", passed ? "." : "X");
    if (!passed) {
        puts("\ntest_compact: FAILED");
    }

    puts("");
    return all_passed ? EXIT_SUCCESS : EXIT_FAILURE;
}

int test_find_context_dups() {
    const size_t vec_size = 1234;
    const size_t batch_size = 8;

    // Reference to the first unique vector
    const std::vector<int> shared_contexts_ref{0, 0, 2, 3, 4, 4, 3, 3};

    // Which compact index belong to what vector
    const std::vector<int> batch_idx_to_compact_idx{0, 0, 1, 2, 3, 3, 2, 2};
    std::vector<int> batch_idx_to_compact_idx_test(batch_size);

    // Reverse map of batch_idx_to_compact_idx
    const std::vector<int> compact_idx_to_batch_idx{0, 2, 3, 4, -1, -1, -1, -1};
    std::vector<int> compact_idx_to_batch_idx_test(batch_size, -1);

    std::vector<int> input_ids;
    std::vector<int> default_vector(vec_size, 0);

    for (size_t i = 0; i < batch_size; ++i) {
        default_vector[vec_size - 1] = shared_contexts_ref[i];
        input_ids.insert(input_ids.end(), default_vector.begin(), default_vector.end());
    }

    std::vector<int> shared_contexts_test(batch_size);

    int *d_input_ids;
    int *d_shared_contexts_test;
    int *d_batch_idx_to_compact_idx;
    int *d_compact_to_batch;
    int *d_compact_size;
    hipMalloc(&d_input_ids, batch_size * vec_size * sizeof(int));
    hipMalloc(&d_shared_contexts_test, batch_size * sizeof(int));
    hipMalloc(&d_batch_idx_to_compact_idx, batch_size * sizeof(int));
    hipMalloc(&d_compact_size, sizeof(int));

    cudaH2Dcpy(d_input_ids, input_ids.data(), batch_size * vec_size);
    /*
    invokeFIndContextDups(d_shared_contexts_test,
                          d_batch_idx_to_compact_idx,
                          d_compact_to_batch,
                          d_compact_size,
                          batch_size,
                          1, // beam_width
                          vec_size);
    */

    int compact_size;
    cudaD2Hcpy(shared_contexts_test.data(), d_shared_contexts_test, batch_size);
    cudaD2Hcpy(batch_idx_to_compact_idx_test.data(), d_batch_idx_to_compact_idx, batch_size);
    cudaD2Hcpy(compact_idx_to_batch_idx_test.data(), d_compact_to_batch, batch_size);
    cudaD2Hcpy(&compact_size, d_compact_size, 1);

    hipFree(d_input_ids);
    hipFree(d_shared_contexts_test);

    EXPECT_TRUE(shared_contexts_test == shared_contexts_ref);
    EXPECT_TRUE(batch_idx_to_compact_idx == batch_idx_to_compact_idx_test);
    EXPECT_TRUE(compact_idx_to_batch_idx_test == compact_idx_to_batch_idx);
    EXPECT_TRUE(compact_size == 4);

    return EXIT_SUCCESS;
}

int test_compact() {
    size_t batch_size = 128;
    size_t compact_size = 5;
    size_t seq_len = 40;
    size_t hidden_dimension = 8;

    auto generator_f = std::bind(std::uniform_real_distribution<float>(-1.0, 1.0), std::mt19937());
    auto generator_i = std::bind(std::uniform_int_distribution<int>(0, 128), std::mt19937());

    // decoder_input [batch_size, seq_len, hidden_dimention] ->
    // compact_decoder_input [compact_size, seq_len, hidden_dimension]
    std::vector<float> decoder_input(batch_size * seq_len * hidden_dimension);
    std::vector<float> compact_decoder_input(compact_size * seq_len * hidden_dimension);
    std::generate(decoder_input.begin(), decoder_input.end(), generator_f);
    float *d_decoder_input, *d_compact_decoder_input;
    hipMalloc(&d_decoder_input, decoder_input.size() * sizeof(float));
    hipMalloc(&d_compact_decoder_input, compact_decoder_input.size() * sizeof(float));
    cudaH2Dcpy(d_decoder_input, decoder_input.data(), decoder_input.size());

    // attention_mask [batch_size, seq_len, seq_len] ->
    // compact_attention_mask [compact_size, seq_len, seq_len]
    std::vector<float> attention_mask(batch_size * seq_len * seq_len);
    std::vector<float> compact_attention_mask(compact_size * seq_len * seq_len);
    std::generate(attention_mask.begin(), attention_mask.end(), generator_f);
    float *d_attention_mask, *d_compact_attention_mask;
    hipMalloc(&d_attention_mask, attention_mask.size() * sizeof(float));
    hipMalloc(&d_compact_attention_mask, compact_decoder_input.size() * sizeof(float));
    cudaH2Dcpy(d_attention_mask, attention_mask.data(), attention_mask.size());

    // input_lengths [batch_size] -> compact_input_lengths [compact_size]
    std::vector<int> input_lengths(batch_size);
    std::vector<int> compact_input_lengths(compact_size);
    std::generate(input_lengths.begin(), input_lengths.end(), generator_i);
    int *d_input_lengths, *d_compact_input_lengths;
    hipMalloc(&d_input_lengths, input_lengths.size() * sizeof(int));
    hipMalloc(&d_compact_input_lengths, compact_input_lengths.size() * sizeof(int));
    cudaH2Dcpy(d_input_lengths, input_lengths.data(), input_lengths.size());

    // compact_idx [compact_size]
    std::vector<int> compact_idx{0, 29, 42, 44, 100};
    int *d_compact_idx;
    hipMalloc(&d_compact_idx, compact_idx.size() * sizeof(int));
    cudaH2Dcpy(d_compact_idx, compact_idx.data(), compact_idx.size());

    invokeCompactInputs<float>(d_compact_decoder_input,
                               d_compact_attention_mask,
                               d_compact_input_lengths,
                               d_decoder_input,
                               d_attention_mask,
                               d_input_lengths,
                               d_compact_idx,
                               compact_size,
                               seq_len,
                               hidden_dimension);

    cudaD2Hcpy(compact_decoder_input.data(), d_compact_decoder_input, compact_size * seq_len * hidden_dimension);
    cudaD2Hcpy(compact_attention_mask.data(), d_compact_attention_mask, compact_size * seq_len * seq_len);
    cudaD2Hcpy(compact_input_lengths.data(), d_compact_input_lengths, compact_size);

    for (size_t i = 0; i < compact_size; ++i) {
        for (size_t t = 0; t < seq_len; ++t) {
            for (size_t h = 0; h < hidden_dimension; ++h) {
                EXPECT_TRUE(compact_decoder_input[(i * seq_len + t) * hidden_dimension + h]
                            == decoder_input[(compact_idx[i] * seq_len + t) * hidden_dimension + h]);
            }
        }
    }

    for (size_t i = 0; i < compact_size; ++i) {
        for (size_t t1 = 0; t1 < seq_len; ++t1) {
            for (size_t t2 = 0; t2 < seq_len; ++t2) {
                EXPECT_TRUE(compact_attention_mask[(i * seq_len + t1) * seq_len + t2]
                            == attention_mask[(compact_idx[i] * seq_len + t1) * seq_len + t2]);
            }
        }
    }

    for (size_t i = 0; i < compact_size; ++i) {
        EXPECT_TRUE(compact_input_lengths[i] == input_lengths[compact_idx[i]]);
    }

    hipFree(d_decoder_input);
    hipFree(d_compact_decoder_input);
    hipFree(d_attention_mask);
    hipFree(d_compact_attention_mask);
    hipFree(d_input_lengths);
    hipFree(d_compact_input_lengths);
    hipFree(d_compact_idx);

    return EXIT_SUCCESS;
}
