#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <math.h>
#include <vector>
#include <tuple>
#include <string>

#include "utils/logger.h"
#include "utils/memory_utils.h"
#include "utils/allocator.h"
#include "utils/cuda_utils.h"
#include "utils/cublasMMWrapper.h"
#include "utils/gemm.h"

using namespace space_llm;

class TestFailureError : public std::exception {
private:
    std::string msg_;

public:
    explicit TestFailureError() = default;
    explicit TestFailureError(std::string name, std::string msg = "") {
        msg_ = fmtstr("TEST FAIL [%s] %s", name.c_str(), msg.c_str());
    }
    const char *what() const throw() {
        return msg_.c_str();
    }
};

#define EXPECT_ALMOST_EQUAL(name, dtype, ctype, out, ref)       \
    do {                                                        \
        bool is_ok = checkResult<dtype, ctype>(name, out, ref); \
        if (!is_ok) {                                           \
            QK_LOG_ERROR("TEST FAIL [%s] at %s:%d",             \
                         __func__, __FILE__, __LINE__);         \
            throw TestFailureError(__func__);                   \
        }                                                       \
    } while (false)

// TensorWrapper is to handle a tensor object as well as its memory buffer,
// because tensor.data is const we cannot set values.
class TensorWrapper {
private:
    IAllocator *allocator;

public:
    std::vector<size_t> shape;
    DataType type;
    Tensor *tensor;
    void *data;

    TensorWrapper(IAllocator *allocator, DataType dtype, std::vector<size_t> shape, bool zero_init = false) {
        this->allocator = allocator;
        this->type = dtype;
        this->shape = shape;

        size_t tensor_memsize = this->memsize();
        this->data = this->allocator->malloc(tensor_memsize, false);

        if (zero_init) {
            check_cuda_error(hipMemset(data, 0x0, tensor_memsize));
        } else {
            setRandomValues();
        }
        this->tensor = new Tensor(MEMORY_GPU, dtype, shape, data);
    }

    TensorWrapper(TensorWrapper const &other) :
        allocator(other.allocator), shape(other.shape), type(other.type), data(other.data), tensor(other.tensor) {
        QK_LOG_DEBUG("TensorWrapper copy: this=%p other=%p", data, other.data);
    }

    ~TensorWrapper() {
        delete tensor;
        allocator->free((void **)&data);
    }

    void setInvalidValues() {
        size_t type_size = (tensor->type == TYPE_FP16) ? sizeof(float) : sizeof(half);
        size_t tensor_size = type_size * tensor->size();
        // Fill by a random number to guarantee invalid values
        check_cuda_error(hipMemset(data, 0xdc, tensor_size));
    }

    void setRandomValues() {
        // random initialization
        size_t num_elements = this->size();
        switch (this->type) {
        case TYPE_FP32:
            cudaRandomUniform((float *)data, num_elements);
            break;
        case TYPE_FP16:
            cudaRandomUniform((half *)data, num_elements);
            break;
        default:
            throw std::runtime_error("Not supported data type");
        }
    }

    size_t size() {
        size_t n_elements = 1;
        for (size_t s : this->shape) {
            n_elements *= s;
        }
        return n_elements;
    }

    size_t memsize() {
        size_t type_size = 0;
        switch (this->type) {
        case TYPE_FP32:
            type_size = sizeof(float);
            break;
        case TYPE_FP16:
            type_size = sizeof(half);
            break;
        default:
            throw std::runtime_error("Not supported data type");
        }
        return type_size * this->size();
    }
};

template <DataType computeType>
void computeReference(GemmOp transa, GemmOp transb,
                      TensorWrapper &C, TensorWrapper &A, TensorWrapper &B,
                      float alpha = 1.0f, float beta = 0.0f) {
    size_t m = C.shape[0];
    size_t n = C.shape[1];
    size_t k = A.shape[1];

    size_t lda = (transa == GEMM_OP_N) ? k : m;
    size_t ldb = (transb == GEMM_OP_N) ? n : k;
    size_t ldc = n;

    hipDataType atype = (A.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType btype = (B.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType ctype = (C.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType compute_type = (computeType == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;

    hipblasHandle_t cublas_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));

    half h_alpha = (half)alpha;
    half h_beta = (half)beta;
    const void *_alpha = (computeType == TYPE_FP16) ? (const void *)&h_alpha : (const void *)&alpha;
    const void *_beta = (computeType == TYPE_FP16) ? (const void *)&h_beta : (const void *)&beta;

    check_cuda_error(hipblasGemmEx(cublas_handle,
                                  getCublasOperation(transa),
                                  getCublasOperation(transb),
                                  n, m, k,
                                  _alpha,
                                  (const void *)B.data, btype, ldb,
                                  (const void *)A.data, atype, lda,
                                  _beta,
                                  (void *)C.data, ctype, ldc,
                                  compute_type,
                                  HIPBLAS_GEMM_DEFAULT));
    check_cuda_error(hipblasDestroy(cublas_handle));
    hipDeviceSynchronize();
}

bool almostEqual(float a, float b, float atol = 1e-5, float rtol = 1e-8) {
    // Params: a = value to compare and b = reference
    // This function follows implementation of numpy.isclose(), which checks
    //   abs(a - b) <= (atol + rtol * abs(b)).
    // Note that the inequality above is asymmetric where b is considered as
    // a reference value. To account into both absolute/relative errors, it
    // uses absolute tolerance and relative tolerance at the same time. The
    // default values of atol and rtol borrowed from numpy.isclose(). For the
    // case of nan value, the result will be true.
    if (isnan(a) && isnan(b)) {
        return true;
    }
    return fabs(a - b) <= (atol + rtol * fabs(b));
}

template <typename T>
bool _checkResult(std::string name, TensorWrapper &out, TensorWrapper &ref, float atol, float rtol) {
    assert(out.type == ref.type);

    size_t out_size = out.size();
    size_t ref_size = ref.size();
    T *h_out = reinterpret_cast<T *>(malloc(sizeof(T) * out_size));
    T *h_ref = reinterpret_cast<T *>(malloc(sizeof(T) * ref_size));

    hipMemcpy(h_out, out.data, sizeof(T) * out_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_ref, ref.data, sizeof(T) * ref_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    size_t failures = 0;
    for (size_t i = 0; i < out_size; ++i) {
        // The values for the output and the reference.
        float a = (float)h_out[i];
        float b = (float)h_ref[i];

        bool ok = almostEqual(a, b, atol, rtol);
        // Print the error.
        if (!ok && failures < 4) {
            QK_LOG_ERROR(">> invalid result for i=%lu:", i);
            QK_LOG_ERROR(">>    found......: %10.6f", a);
            QK_LOG_ERROR(">>    expected...: %10.6f", b);
            QK_LOG_ERROR(">>    error......: %.6f", fabsf(a - b));
            QK_LOG_ERROR(">>    tol........: %.6f", atol + rtol * fabs(b));
        }

        // Update the number of failures.
        failures += ok ? 0 : 1;
    }

    // Allow not matched up to 1% elements.
    size_t tol_failures = (size_t)(0.01 * out_size);
    QK_LOG_INFO("check....... %30s : %s (failures: %.2f%% atol: %.2e rtol: %.2e)",
                name.c_str(), failures <= tol_failures ? "OK" : "FAILED",
                100. * failures / out_size, atol, rtol);
    return failures <= tol_failures;
}

template <typename T, DataType computeType>
bool checkResult(std::string name, TensorWrapper &out, TensorWrapper &ref) {
    float atol = (computeType == TYPE_FP32) ? 1e-6f : 1e-3f;
    float rtol = (computeType == TYPE_FP32) ? 1e-4f : 1e-1f;
    bool is_ok = false;
    if (sizeof(T) == 4) {
        is_ok = _checkResult<float>(name, out, ref, atol, rtol);
    } else {
        is_ok = _checkResult<half>(name, out, ref, atol, rtol);
    }
    return is_ok;
}

template <typename T, DataType computeType>
bool checkResult(TensorWrapper &out, TensorWrapper &ref) {
    return checkResult<T, computeType>("", out, ref);
}

template <typename T>
std::string toString() {
    std::string str = "dtype=";
    str += std::is_same<T, float>::value ? "FP32" : "FP16";
    return str;
}

template <typename T, DataType ctype>
std::string toString() {
    std::string str = "dtype=";
    str += std::is_same<T, float>::value ? "FP32" : "FP16";
    str += ", compute_type=";
    str += (ctype == TYPE_FP32) ? "FP32" : "FP16";
    return str;
}

struct GemmOpPair {
    GemmOp transa;
    GemmOp transb;
};

static const std::vector<GemmOpPair> op_pairs{{GEMM_OP_N, GEMM_OP_N},
                                              {GEMM_OP_N, GEMM_OP_T},
                                              {GEMM_OP_T, GEMM_OP_N},
                                              {GEMM_OP_T, GEMM_OP_T}};

static inline std::string getTestName(const char *func_name, GemmOp transa, GemmOp transb,
                                      size_t m, size_t n, size_t k) {
    return fmtstr("%s [opA=%s, opB=%s, m=%ld, n=%ld, k=%ld]",
                  func_name, getGemmOpString(transa).c_str(), getGemmOpString(transb).c_str(),
                  m, n, k);
}

static inline std::string getTestName(const char *func_name, GemmOpPair op_pairs, size_t m, size_t n, size_t k) {
    return getTestName(func_name, op_pairs.transa, op_pairs.transb, m, n, k);
}

/////////////////////////////////// Unittests //////////////////////////////////////////

template <typename T, DataType computeType>
void testGemmCorrectnessMatmul(size_t m, size_t n, size_t k) {
    QK_LOG_INFO("Matmul function correctness test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    Allocator allocator(getDevice());

    DataType dtype = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, dtype, {m, k}, false);
    TensorWrapper b_tensor(&allocator, dtype, {k, n}, false);
    TensorWrapper c_tensor(&allocator, dtype, {m, n}, true);
    TensorWrapper expected(&allocator, dtype, {m, n}, true);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, false, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);
        QK_LOG_DEBUG(tc_name);
        computeReference<computeType>(op_pair.transa, op_pair.transb,
                                      expected, a_tensor, b_tensor);

        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;

        c_tensor.setInvalidValues(); // to guarantee C has invalid data
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, a_tensor.type, lda,
                   b_tensor.data, b_tensor.type, ldb,
                   c_tensor.data, c_tensor.type, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);
    }
}

int main(int argc, char *argv[]) {
    // testGemmCreate();
    using testcase_t = std::tuple<size_t, size_t, size_t>;

    std::vector<testcase_t> testcases = {{16, 32, 64},
                                         {255, 255, 255},
                                         {1041, 2047, 9999},
                                         {1041, 1, 9999},
                                         {1041, 999, 1}};

    // Computation correctness tests
    for (testcase_t &tc : testcases) {
        size_t m = std::get<0>(tc);
        size_t n = std::get<1>(tc);
        size_t k = std::get<2>(tc);

        testGemmCorrectnessMatmul<float, TYPE_FP32>(m, n, k);
    }

    return 0;
}
