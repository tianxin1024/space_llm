#include "hip/hip_runtime.h"
#include "kernels/layernorm_kernels.h"
#include "kernels/reduce_kernel_utils.cuh"
#include "utils/cuda_type_utils.cuh"
#include <hip/hip_fp16.h>
#include <math.h>

namespace space_llm {

// * Note that typename T is half2 or bfloat2 type
template <typename T, bool IS_OUTPUT, bool IS_BIAS, int RESIDUAL_NUM, bool IS_BETA, int UNROLL_FACTOR>
__global__ void generalAddBiasResidualLayerNormOpt(T *normed_output,
                                                   T *output,
                                                   const T *__restrict input,
                                                   const T *__restrict bias,
                                                   const T *__restrict residual1,
                                                   const T *__restrict residual2,
                                                   const T *__restrict gamma,
                                                   const T *__restrict beta,
                                                   const float layernorm_eps,
                                                   int m,
                                                   int n,
                                                   const float *scale_inter,
                                                   const float *scale_out,
                                                   const float *scale,
                                                   float *dynamic_scale,
                                                   const int int8_mode) {
    extern __shared__ __align__(sizeof(float)) char _shmem[]; // Align on largest type
    T *shmem = reinterpret_cast<T *>(_shmem);

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;   // int16_t
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type; // half2
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;   // float
    using Scalar_T = typename packed_as<T, 1>::type;

    const bool scale_input = int8_mode == 2 && scale_inter != nullptr;
    const bool dynamic_scaling = dynamic_scale != nullptr;

    T local_sum = cuda_cast<T>(0.0f);

    const Float_Packed_T scale_from_int = cuda_cast<Float_Packed_T>(scale_input ? (*scale_inter) * (*scale_out) : 0.0f);
    const Float_Packed_T scale_to_int = cuda_cast<Float_Packed_T>(int8_mode == 2 ? *scale : 0.0f);

#pragma unroll
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        T val = cuda_cast<T>(0.0f);

        if (IS_BIAS) {
            val = hadd2(val, ldg(&bias[i]));
        }
        if (RESIDUAL_NUM == 1) {
            val = hadd2(val, ldg(&residual1[index]));
        } else if (RESIDUAL_NUM == 2) {
            val = hadd2(hadd2(val, ldg(&residual1[index])), ldg(&residual2[index]));
        }

        if (IS_OUTPUT) {
            T in_val;
            if (scale_input) {
                in_val = cuda_cast<T>(cuda_cast<Float_Packed_T>(reinterpret_cast<const Int32_Packed_T *>(input)[index])
                                      * scale_from_int);
            } else {
                in_val = input[index];
            }
            val = hadd2(val, in_val);
        }
        shmem[i] = val;
        output[index] = val;
        local_sum = hadd2(local_sum, val);
    }

    mean = blockReduceSum((float)(local_sum.x + local_sum.y));

    if (threadIdx.x == 0) {
        s_mean = mean / n / 2;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        T val = input[blockIdx.x * n + i];
        float diff_1 = (float)(val.x) - s_mean;
        float diff_2 = (float)(val.y) - s_mean;
        local_var_sum += (diff_1 * diff_1 + diff_2 * diff_2);
    }

    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n / 2 + layernorm_eps);
    }
    __syncthreads();

    T mean_2 = cuda_cast<T>(s_mean);
    T var_2 = cuda_cast<T>(s_variance);

    Scalar_T abs_max = 1e-6f;

#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        T val = hmul2(hsub2(shmem[i], mean_2), var_2, ldg(&gamma[i]));
        if (IS_BETA) {
            val = hadd2(val, ldg(&beta[i]));
        }

        if (dynamic_scaling) {
            abs_max = cuda_max(cuda_max<Scalar_T>(cuda_abs(val)), abs_max);
            shmem[i] = val;
        } else if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(val) * scale_to_int);
        } else {
            normed_output[index] = val;
        }
    }

    if (dynamic_scaling) {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(abs_max));
        const float dynamic_per_token_scale = 127. / abs_max_f;
        for (int i = threadIdx.x; i < n; i += blockDim.x) {
            const int index = blockIdx.x * n + i;
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] = cuda_cast<Int8_Packed_T>(
                cuda_cast<Float_Packed_T>(shmem[i]) * cuda_cast<Float_Packed_T>(dynamic_per_token_scale));
        }
        if (threadIdx.x == 0) {
            dynamic_scale[blockIdx.x] = (*scale * abs_max_f) / 127.f;
        }
    }
}

// * Note that typename T is half2 or bfloat2 type
template <typename T, bool IS_OUTPUT, bool IS_BIAS, int RESIDUAL_NUM, bool IS_BETA, int UNROLL_FACTOR>
__global__ void generalAddBiasResidualLayerNormOpt2(T *normed_output,
                                                    T *output,
                                                    const T *__restrict input,
                                                    const T *__restrict bias,
                                                    const T *__restrict residual1,
                                                    const T *__restrict residual2,
                                                    const T *__restrict gamma,
                                                    const T *__restrict beta,
                                                    const float layernorm_eps,
                                                    int m,
                                                    int n,
                                                    const float *scale_inter,
                                                    const float *scale_out,
                                                    const float *scale,
                                                    float *dynamic_scale,
                                                    const int int8_mode) {
    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T *shmem = reinterpret_cast<T *>(_shmem);

    __shared__ float s_mean;
    __shared__ float s_variance;
    float x_sum = 0.0f;
    float x2_sum = 0.0f;
    const int b_offset = blockIdx.x * n;

    using T1 = typename TypeConverter<T>::Type;
    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    using Scalar_T = typename packed_as<T, 1>::type;

    const bool scale_input = int8_mode == 2 && scale_inter != nullptr;
    const Float_Packed_T scale_vec_in = cuda_cast<Float_Packed_T>(scale_input ? (*scale_inter) * (*scale_out) : 0.0f);
    const Float_Packed_T scale_vec = cuda_cast<Float_Packed_T>(int8_mode == 2 ? *scale : 0.0f);
    const bool dynamic_scaling = dynamic_scale != nullptr;

#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = b_offset + i;
        float val_1 = 0.0f;
        float val_2 = 0.0f;
        T tmp;

        if (IS_BIAS) {
            tmp = ldg(&bias[i]);
            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        }
        if (RESIDUAL_NUM == 1) {
            tmp = ldg(&residual1[index]);
            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        } else if (RESIDUAL_NUM == 2) {
            tmp = ldg(&residual1[index]);
            T tmp2 = ldg(&residual2[index]);
            val_1 += (static_cast<float>(tmp.x) + static_cast<float>(tmp2.x));
            val_2 += (static_cast<float>(tmp.y) + static_cast<float>(tmp2.y));
        }

        if (IS_OUTPUT) {
            if (scale_input) {
                tmp = cuda_cast<T>(cuda_cast<Float_Packed_T>(reinterpret_cast<const Int32_Packed_T *>(input)[index])
                                   * scale_vec_in);
            } else {
                tmp = ldg(&input[index]);
            }
            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        }
        tmp.x = cuda_cast<T1>(val_1);
        tmp.y = cuda_cast<T1>(val_2);
        shmem[i] = tmp;
        output[index] = tmp;
        x_sum += val_1 + val_2;
        x2_sum += val_1 * val_1 + val_2 * val_2;
    }
    float sums[2];
    sums[0] = x_sum;
    sums[1] = x2_sum;
    blockReduceSumV2<float, 2>(sums);

    if (threadIdx.x == 0) {
        s_mean = sums[0] / n / 2;
        s_variance = rsqrtf(sums[1] / n / 2 - s_mean * s_mean + layernorm_eps);
    }
    __syncthreads();

    T mean_2 = cuda_cast<T>(s_mean);
    T var_2 = cuda_cast<T>(s_variance);

    Scalar_T abs_max = 1e-6f;

#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        T val = hmul2(hsub2(shmem[i], mean_2), var_2, ldg(&gamma[i]));
        if (IS_BETA) {
            val = hadd2(val, ldg(&beta[i]));
        }

        if (dynamic_scaling) {
            abs_max = cuda_max(cuda_max<Scalar_T>(cuda_abs(val)), abs_max);
            shmem[i] = val;
        } else if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(val) * scale_vec);
        } else {
            normed_output[index] = val;
        }
    }

    if (dynamic_scaling) {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(abs_max));
        const float dynamic_per_token_scale = 127. / abs_max_f;
        for (int i = threadIdx.x; i < n; i += blockDim.x) {
            const int index = blockIdx.x * n + i;
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] = cuda_cast<Int8_Packed_T>(
                cuda_cast<Float_Packed_T>(shmem[i]) * cuda_cast<Float_Packed_T>(dynamic_per_token_scale));
        }
        if (threadIdx.x == 0) {
            dynamic_scale[blockIdx.x] = (*scale * abs_max_f) / 127.f;
        }
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, int UNROLL_FACTOR, int RESIDUAL_NUM>
void dispatch_generalAddBiasResidualLayerNormOpt_opt_version(T *norm_output,
                                                             T *output,
                                                             const T *input,
                                                             const T *bias,
                                                             const T *residual1,
                                                             const T *residual2,
                                                             const T *gamma,
                                                             const T *beta,
                                                             float layernorm_eps,
                                                             int m,
                                                             int half_n,
                                                             const float *scale_inter,
                                                             const float *scale_out,
                                                             const float *scale,
                                                             float *dynamic_scale,
                                                             int int8_mode,
                                                             dim3 grid,
                                                             dim3 block,
                                                             hipStream_t stream,
                                                             int opt_version) {
    size_t maxbytes = half_n * sizeof(T);
    if (opt_version == 1) {
        if (maxbytes >= (48 << 10)) {
            check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                generalAddBiasResidualLayerNormOpt<T), IS_OUTPUT, IS_BIAS, RESIDUAL_NUM, true, UNROLL_FACTOR>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                maxbytes));
        }
        generalAddBiasResidualLayerNormOpt<T, IS_OUTPUT, IS_BIAS, RESIDUAL_NUM, true, UNROLL_FACTOR>
            <<<grid, block, maxbytes, stream>>>(norm_output,
                                                output,
                                                input,
                                                bias,
                                                residual1,
                                                residual2,
                                                gamma,
                                                beta,
                                                layernorm_eps,
                                                m,
                                                half_n,
                                                scale_inter,
                                                scale_out,
                                                scale,
                                                dynamic_scale,
                                                int8_mode);
    } else if (opt_version == 2) {
        if (maxbytes >= (48 << 10)) {
            check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                generalAddBiasResidualLayerNormOpt2<T), IS_OUTPUT, IS_BIAS, RESIDUAL_NUM, true, UNROLL_FACTOR>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                maxbytes));
        }
        generalAddBiasResidualLayerNormOpt2<T, IS_OUTPUT, IS_BIAS, RESIDUAL_NUM, true, UNROLL_FACTOR>
            <<<grid, block, maxbytes, stream>>>(norm_output,
                                                output,
                                                input,
                                                bias,
                                                residual1,
                                                residual2,
                                                gamma,
                                                beta,
                                                layernorm_eps,
                                                m,
                                                half_n,
                                                scale_inter,
                                                scale_out,
                                                scale,
                                                dynamic_scale,
                                                int8_mode);
    } else {
        QK_CHECK_WITH_INFO(false, "opt_num must be 1 or 2");
    }
}

template <typename T, bool IS_BIAS, int UNROLL_FACTOR, int RESIDUAL_NUM>
void dispatch_generalAddBiasResidualLayerNormOpt_is_output(T *norm_output,
                                                           T *output,
                                                           const T *input,
                                                           const T *bias,
                                                           const T *residual1,
                                                           const T *residual2,
                                                           const T *gamma,
                                                           const T *beta,
                                                           float layernorm_eps,
                                                           int m,
                                                           int half_n,
                                                           const float *scale_inter,
                                                           const float *scale_out,
                                                           const float *scale,
                                                           float *dynamic_scale,
                                                           int int8_mode,
                                                           dim3 grid,
                                                           dim3 block,
                                                           hipStream_t stream,
                                                           int opt_version,
                                                           bool is_output) {
    if (is_output) {
        dispatch_generalAddBiasResidualLayerNormOpt_opt_version<T, true, IS_BIAS, UNROLL_FACTOR, RESIDUAL_NUM>(
            norm_output,
            output,
            input,
            bias,
            residual1,
            residual2,
            gamma,
            beta,
            layernorm_eps,
            m,
            half_n,
            scale_inter,
            scale_out,
            scale,
            dynamic_scale,
            int8_mode,
            grid,
            block,
            stream,
            opt_version);
    } else {
        dispatch_generalAddBiasResidualLayerNormOpt_opt_version<T, false, IS_BIAS, UNROLL_FACTOR, RESIDUAL_NUM>(
            norm_output,
            output,
            input,
            bias,
            residual1,
            residual2,
            gamma,
            beta,
            layernorm_eps,
            m,
            half_n,
            scale_inter,
            scale_out,
            scale,
            dynamic_scale,
            int8_mode,
            grid,
            block,
            stream,
            opt_version);
    }
}

template <typename T, int UNROLL_FACTOR, int RESIDUAL_NUM>
void dispatch_generalAddBiasResidualLayerNormOpt_bias(T *norm_output,
                                                      T *output,
                                                      const T *input,
                                                      const T *bias,
                                                      const T *residual1,
                                                      const T *residual2,
                                                      const T *gamma,
                                                      const T *beta,
                                                      float layernorm_eps,
                                                      int m,
                                                      int half_n,
                                                      const float *scale_inter,
                                                      const float *scale_out,
                                                      const float *scale,
                                                      float *dynamic_scale,
                                                      int int8_mode,
                                                      dim3 grid,
                                                      dim3 block,
                                                      hipStream_t stream,
                                                      int opt_version,
                                                      bool is_output) {
    if (bias != nullptr) {
        dispatch_generalAddBiasResidualLayerNormOpt_is_output<T, true, UNROLL_FACTOR, RESIDUAL_NUM>(norm_output,
                                                                                                    output,
                                                                                                    input,
                                                                                                    bias,
                                                                                                    residual1,
                                                                                                    residual2,
                                                                                                    gamma,
                                                                                                    beta,
                                                                                                    layernorm_eps,
                                                                                                    m,
                                                                                                    half_n,
                                                                                                    scale_inter,
                                                                                                    scale_out,
                                                                                                    scale,
                                                                                                    dynamic_scale,
                                                                                                    int8_mode,
                                                                                                    grid,
                                                                                                    block,
                                                                                                    stream,
                                                                                                    opt_version,
                                                                                                    is_output);
    } else {
        dispatch_generalAddBiasResidualLayerNormOpt_is_output<T, false, UNROLL_FACTOR, RESIDUAL_NUM>(norm_output,
                                                                                                     output,
                                                                                                     input,
                                                                                                     bias,
                                                                                                     residual1,
                                                                                                     residual2,
                                                                                                     gamma,
                                                                                                     beta,
                                                                                                     layernorm_eps,
                                                                                                     m,
                                                                                                     half_n,
                                                                                                     scale_inter,
                                                                                                     scale_out,
                                                                                                     scale,
                                                                                                     dynamic_scale,
                                                                                                     int8_mode,
                                                                                                     grid,
                                                                                                     block,
                                                                                                     stream,
                                                                                                     opt_version,
                                                                                                     is_output);
    }
}

template <typename T, int UNROLL_FACTOR>
void dispatch_generalAddBiasResidualLayerNormOpt_residual_num(T *norm_output,
                                                              T *output,
                                                              const T *input,
                                                              const T *bias,
                                                              const T *residual1,
                                                              const T *residual2,
                                                              const T *gamma,
                                                              const T *beta,
                                                              float layernorm_eps,
                                                              int m,
                                                              int half_n,
                                                              const float *scale_inter,
                                                              const float *scale_out,
                                                              const float *scale,
                                                              float *dynamic_scale,
                                                              int int8_mode,
                                                              dim3 grid,
                                                              dim3 block,
                                                              hipStream_t stream,
                                                              int opt_version,
                                                              bool is_output,
                                                              int residual_num) {
    if (residual_num == 1) {
        dispatch_generalAddBiasResidualLayerNormOpt_bias<T, UNROLL_FACTOR, 1>(norm_output,
                                                                              output,
                                                                              input,
                                                                              bias,
                                                                              residual1,
                                                                              residual2,
                                                                              gamma,
                                                                              beta,
                                                                              layernorm_eps,
                                                                              m,
                                                                              half_n,
                                                                              scale_inter,
                                                                              scale_out,
                                                                              scale,
                                                                              dynamic_scale,
                                                                              int8_mode,
                                                                              grid,
                                                                              block,
                                                                              stream,
                                                                              opt_version,
                                                                              is_output);
    } else if (residual_num == 2) {
        dispatch_generalAddBiasResidualLayerNormOpt_bias<T, UNROLL_FACTOR, 2>(norm_output,
                                                                              output,
                                                                              input,
                                                                              bias,
                                                                              residual1,
                                                                              residual2,
                                                                              gamma,
                                                                              beta,
                                                                              layernorm_eps,
                                                                              m,
                                                                              half_n,
                                                                              scale_inter,
                                                                              scale_out,
                                                                              scale,
                                                                              dynamic_scale,
                                                                              int8_mode,
                                                                              grid,
                                                                              block,
                                                                              stream,
                                                                              opt_version,
                                                                              is_output);
    } else {
        QK_CHECK_WITH_INFO(false, "residual_num must be 1 or 2");
    }
}

template <typename T>
void dispatch_generalAddBiasResidualLayerNormOpt_unroll_factor(T *norm_output,
                                                               T *output,
                                                               const T *input,
                                                               const T *bias,
                                                               const T *residual1,
                                                               const T *residual2,
                                                               const T *gamma,
                                                               const T *beta,
                                                               float layernorm_eps,
                                                               int m,
                                                               int half_n,
                                                               const float *scale_inter,
                                                               const float *scale_out,
                                                               const float *scale,
                                                               float *dynamic_scale,
                                                               int int8_mode,
                                                               dim3 grid,
                                                               dim3 block,
                                                               hipStream_t stream,
                                                               int opt_version,
                                                               bool is_output,
                                                               int residual_num,
                                                               int unroll_factor) {
    switch (unroll_factor) {
    case 1:
        dispatch_generalAddBiasResidualLayerNormOpt_residual_num<T, 1>(norm_output,
                                                                       output,
                                                                       input,
                                                                       bias,
                                                                       residual1,
                                                                       residual2,
                                                                       gamma,
                                                                       beta,
                                                                       layernorm_eps,
                                                                       m,
                                                                       half_n,
                                                                       scale_inter,
                                                                       scale_out,
                                                                       scale,
                                                                       dynamic_scale,
                                                                       int8_mode,
                                                                       grid,
                                                                       block,
                                                                       stream,
                                                                       opt_version,
                                                                       is_output,
                                                                       residual_num);
        break;
    case 2:
        dispatch_generalAddBiasResidualLayerNormOpt_residual_num<T, 2>(norm_output,
                                                                       output,
                                                                       input,
                                                                       bias,
                                                                       residual1,
                                                                       residual2,
                                                                       gamma,
                                                                       beta,
                                                                       layernorm_eps,
                                                                       m,
                                                                       half_n,
                                                                       scale_inter,
                                                                       scale_out,
                                                                       scale,
                                                                       dynamic_scale,
                                                                       int8_mode,
                                                                       grid,
                                                                       block,
                                                                       stream,
                                                                       opt_version,
                                                                       is_output,
                                                                       residual_num);
        break;
    case 4:
        dispatch_generalAddBiasResidualLayerNormOpt_residual_num<T, 4>(norm_output,
                                                                       output,
                                                                       input,
                                                                       bias,
                                                                       residual1,
                                                                       residual2,
                                                                       gamma,
                                                                       beta,
                                                                       layernorm_eps,
                                                                       m,
                                                                       half_n,
                                                                       scale_inter,
                                                                       scale_out,
                                                                       scale,
                                                                       dynamic_scale,
                                                                       int8_mode,
                                                                       grid,
                                                                       block,
                                                                       stream,
                                                                       opt_version,
                                                                       is_output,
                                                                       residual_num);
        break;
    case 8:
        dispatch_generalAddBiasResidualLayerNormOpt_residual_num<T, 8>(norm_output,
                                                                       output,
                                                                       input,
                                                                       bias,
                                                                       residual1,
                                                                       residual2,
                                                                       gamma,
                                                                       beta,
                                                                       layernorm_eps,
                                                                       m,
                                                                       half_n,
                                                                       scale_inter,
                                                                       scale_out,
                                                                       scale,
                                                                       dynamic_scale,
                                                                       int8_mode,
                                                                       grid,
                                                                       block,
                                                                       stream,
                                                                       opt_version,
                                                                       is_output,
                                                                       residual_num);
        break;
    default:
        QK_CHECK_WITH_INFO(false, "unroll_factor must be 1, 2, 4 or 8");
    }
}

template <typename T, bool DYNAMIC_SCALING = false>
__global__ void generalLayerNorm(const T *__restrict input,
                                 const T *__restrict gamma,
                                 const T *__restrict beta,
                                 T *normed_output,
                                 const float layernorm_eps,
                                 int m,
                                 int n,
                                 float *scale,
                                 float *dynamic_scale,
                                 const int int8_mode) {
    const int tid = threadIdx.x;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T *shmem = reinterpret_cast<T *>(_shmem);

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    using Scalar_T = typename packed_as<T, 1>::type;

    const Float_Packed_T scale_to_int = cuda_cast<Float_Packed_T>(int8_mode == 2 ? *scale : 0.0f);

    float local_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        local_sum += (float)(ldg(&input[blockIdx.x * n + i]));
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }

    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(ldg(&input[blockIdx.x * n + i])) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

    Scalar_T abs_max = 1e-6f;

    for (int i = tid; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        float beta_val = (beta == nullptr) ? 0.0f : (float)ldg(&beta[i]);
        T val = (T)((((float)input[index] - s_mean) * s_variance) * (float)(ldg(&gamma[i])) + beta_val);

        if (DYNAMIC_SCALING) {
            abs_max = cuda_max(cuda_max<Scalar_T, T>(cuda_abs(val)), abs_max);
            shmem[i] = val;
        } else if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(val) * scale_to_int);
        } else {
            normed_output[index] = val;
        }
    }

    if (DYNAMIC_SCALING) {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(abs_max));
        const Scalar_T dynamic_per_token_scale = 127. / abs_max_f;
        for (int i = tid; i < n; i += blockDim.x) {
            const int index = blockIdx.x * n + i;
            reinterpret_cast<Int8_Packed_T *>(normed_output)[index] = cuda_cast<Int8_Packed_T>(
                cuda_cast<Float_Packed_T>(shmem[i]) * cuda_cast<Float_Packed_T>(dynamic_per_token_scale));
        }
        if (threadIdx.x == 0) {
            dynamic_scale[blockIdx.x] = (*scale * abs_max_f) / 127.f;
        }
    }
}

template <typename T>
void invokeGeneralLayerNorm(T *out,
                            const T *input,
                            const T *gamma,
                            const T *beta,
                            const float layernorm_eps,
                            const int m,
                            const int n,
                            float *scale,
                            float *dynamic_scale,
                            const int int8_mode,
                            hipStream_t stream,
                            int opt_version) {
    dim3 grid(m);
    const bool dynamic_quant = dynamic_scale != nullptr;
#ifdef ENABLE_BF16
    if (n % 2 == 0 && (std::is_save<T, half>::value || std::is_same<T, __hip_bfloat16>::value)
#else
    if (n % 2 == 0 && (std::is_same<T, half>::value)
#endif
        && opt_version > 0) {

        int half_n = n / 2;
        int half_n_32 = (half_n + 31) / 32 * 32;
        dim3 block(std::min(half_n_32, 512));
        int rolls_per_thread = half_n / block.x;
        int unroll_factor = 8;
        while (unroll_factor > rolls_per_thread && unroll_factor > 1) {
            unroll_factor /= 2;
        }
        using T2 = typename TypeConverter<T>::Type;

        /* we launch (and instantiate) the kernel by specializing for unroll_factor -> residual_num -> is_bias ->
         * opt_version */
        dispatch_generalAddBiasResidualLayerNormOpt_unroll_factor((T2 *)out,
                                                                  (T2 *)out,
                                                                  (const T2 *)out,
                                                                  (const T2 *)nullptr,
                                                                  (const T2 *)input,
                                                                  (const T2 *)nullptr,
                                                                  (const T2 *)gamma,
                                                                  (const T2 *)beta,
                                                                  layernorm_eps,
                                                                  m,
                                                                  half_n,
                                                                  nullptr,
                                                                  nullptr,
                                                                  scale,
                                                                  dynamic_scale,
                                                                  int8_mode,
                                                                  grid,
                                                                  block,
                                                                  stream,
                                                                  opt_version,
                                                                  false, // is_output
                                                                  1,     // residual_num
                                                                  unroll_factor);
    } else {
        dim3 block(std::min(n, 1024));

        /* For general cases, n is equal to hidden_units, e.g., 512/1024.
            Since we have warp shuffle inside the code, block.x % 32 should be 0.
        */
        if (n % 32 != 0) {
            block.x = 1024;
        }

        /* should pay attention to the rsqrt precision*/
        if (dynamic_quant) {
            size_t maxbytes = n * sizeof(T);
            if (maxbytes >= (48 << 10)) {
                check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    generalLayerNorm<T), true>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes));
            }
            generalLayerNorm<T, true><<<grid, block, maxbytes, stream>>>(
                input, gamma, beta, out, layernorm_eps, m, n, scale, dynamic_scale, int8_mode); // For gpt-3
        } else {
            generalLayerNorm<T, false><<<grid, block, 0, stream>>>(
                input, gamma, beta, out, layernorm_eps, m, n, scale, dynamic_scale, int8_mode); // For gpt-3
        }
    }
}

template void invokeGeneralLayerNorm(float *out,
                                     const float *input,
                                     const float *gamma,
                                     const float *beta,
                                     const float layernorm_eps,
                                     const int m,
                                     const int n,
                                     float *scale,
                                     float *dynamic_scale,
                                     const int int8_mode,
                                     hipStream_t stream,
                                     int opt_version);

template void invokeGeneralLayerNorm(half *out,
                                     const half *input,
                                     const half *gamma,
                                     const half *beta,
                                     const float layernorm_eps,
                                     const int m,
                                     const int n,
                                     float *scale,
                                     float *dynamic_scale,
                                     const int int8_mode,
                                     hipStream_t stream,
                                     int opt_version);

template <typename T, int RESIDUAL_NUM>
__global__ void generalAddBiasResidualLayerNorm(const T *__restrict input,
                                                const T *__restrict residual1,
                                                const T *__restrict residual2,
                                                const T *__restrict gamma,
                                                const T *__restrict beta,
                                                const T *__restrict bias,
                                                T *output,
                                                T *norm_output,
                                                const float layernorm_eps,
                                                int m,
                                                int n,
                                                const float *scale_inter,
                                                const float *scale_out,
                                                const float *scale,
                                                float *dynamic_scale,
                                                const int int8_mode) {
    int tid = threadIdx.x;

    // NOTE: float shmem may exceed the shared memory limit
    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T *shmem = reinterpret_cast<T *>(_shmem);

    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    using Scalar_T = typename packed_as<T, 1>::type;

    const bool dynamic_scaling = dynamic_scale != nullptr;

    __shared__ float s_mean;
    __shared__ float s_variance;

    float mean = 0.0f;
    float variance = 0.0f;

    const bool is_input_i32 = int8_mode == 2 && scale_inter != nullptr && scale_out != nullptr;
    const float scale_out_val = is_input_i32 ? (*scale_inter) * (*scale_out) : 0.0f;

    float local_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float local_out = 0.0f;
        if (RESIDUAL_NUM == 1) {
            local_out = (float)(ldg(&residual1[blockIdx.x * n + i]));
        } else if (RESIDUAL_NUM == 2) {
            local_out = (float)(ldg(&residual1[blockIdx.x * n + i])) + float(ldg(&residual2[blockIdx.x * n + i]));
        }

        if (is_input_i32) {
            local_out += cuda_cast<float>(reinterpret_cast<const int32_t *>(input)[blockIdx.x * n + i]) * scale_out_val;
        } else {
            local_out += (float)(input[blockIdx.x * n + i]);
        }

        if (bias != nullptr) {
            local_out += (float)(ldg(&bias[i]));
        }
        shmem[i] = (T)local_out;
        output[blockIdx.x * n + i] = (T)local_out;
        local_sum += local_out;
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float diff = (float)(output[blockIdx.x * n + i]) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

    Scalar_T abs_max = 1e-6f;

    const float scale_val = int8_mode == 2 ? *scale : 0.0f;
    for (int i = tid; i < n; i += blockDim.x) {
        float beta_val = (beta == nullptr) ? 0.0f : (float)(ldg(&beta[i]));
        const float val = ((((float)shmem[i] - s_mean) * s_variance) * (float)(ldg(&gamma[i])) + beta_val);

        if (dynamic_scaling) {
            abs_max = cuda_max(cuda_max<Scalar_T, float>(cuda_abs(val)), abs_max);
        } else if (int8_mode == 2) {
            reinterpret_cast<int8_t *>(norm_output)[blockIdx.x * n + i] = cuda_cast<int8_t>(val * scale_val);
        } else {
            norm_output[blockIdx.x * n + i] = (T)val;
        }
    }

    if (dynamic_scaling) {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(abs_max));
        const float dynamic_per_token_scale = 127. / abs_max_f;
        for (int i = tid; i < n; i += blockDim.x) {
            const int index = blockIdx.x * n + i;
            reinterpret_cast<Int8_Packed_T *>(norm_output)[index] = cuda_cast<Int8_Packed_T>(
                cuda_cast<Float_Packed_T>(shmem[i]) * cuda_cast<Float_Packed_T>(dynamic_per_token_scale));
        }
        if (threadIdx.x == 0) {
            dynamic_scale[blockIdx.x] = (*scale * abs_max_f) / 127.f;
        }
    }
}

/*
    output      <- output + bias + residual1 + residual2
    output_norm <- LN(output)
*/
template <typename T>
void invokeGeneralAddBiasResidualPreLayerNorm(T *output,
                                              T *norm_output,
                                              const T *input,
                                              const T *residual1,
                                              const T *residual2,
                                              const T *gamma,
                                              const T *beta,
                                              const T *bias,
                                              const float layernorm_eps,
                                              int m,
                                              int n,
                                              const float *scale_inter,
                                              const float *scale_out,
                                              float *scale,
                                              float *dynamic_scale,
                                              const int int8_mode,
                                              hipStream_t stream,
                                              int opt_version) {
    const int residual_num = (residual2 == nullptr) ? 1 : 2;
    const bool dynamic_quant = dynamic_scale != nullptr;

    if (opt_version > 0 && sizeof(T) == 2 && n % 2 == 0) {
        dim3 grid(m);
        int half_n = n / 2;
        int half_n_32 = (half_n + 31) / 32 * 32;
        dim3 block(std::min(half_n_32, 512));
        int rolls_per_thread = half_n / block.x;
        int unroll_factor = 8;
        while (unroll_factor > rolls_per_thread && unroll_factor > 1) {
            unroll_factor /= 2;
        }

        using T2 = typename TypeConverter<T>::Type;

        /* we launch (and instantiate) the kernel by specializing for unroll_factor -> residual_num -> is_bias ->
         * opt_version */
        dispatch_generalAddBiasResidualLayerNormOpt_unroll_factor((T2 *)norm_output,
                                                                  (T2 *)output,
                                                                  (const T2 *)input,
                                                                  (const T2 *)bias,
                                                                  (const T2 *)residual1,
                                                                  (const T2 *)residual2,
                                                                  (const T2 *)gamma,
                                                                  (const T2 *)beta,
                                                                  layernorm_eps,
                                                                  m,
                                                                  half_n,
                                                                  scale_inter,
                                                                  scale_out,
                                                                  scale,
                                                                  dynamic_scale,
                                                                  int8_mode,
                                                                  grid,
                                                                  block,
                                                                  stream,
                                                                  opt_version,
                                                                  true, // is_output
                                                                  residual_num,
                                                                  unroll_factor);
    } else {
        dim3 grid(m);
        dim3 block(std::min(n, 1024));

        /* For general cases, n is equal to hidden_units, e.g., 512/1024.
        Since we have warp shuffle inside the code, block.x % 32 should be 0.
        */
        block.x = (block.x + 31) / 32 * 32;

        size_t maxbytes = n * sizeof(T);

        if (residual_num == 1) {
            if (maxbytes >= (48 << 10)) {
                check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    generalAddBiasResidualLayerNorm<T), 1>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes));
            }
            generalAddBiasResidualLayerNorm<T, 1><<<grid, block, maxbytes, stream>>>(input,
                                                                                     residual1,
                                                                                     residual2,
                                                                                     gamma,
                                                                                     beta,
                                                                                     bias,
                                                                                     output,
                                                                                     norm_output,
                                                                                     layernorm_eps,
                                                                                     m,
                                                                                     n,
                                                                                     scale_inter,
                                                                                     scale_out,
                                                                                     scale,
                                                                                     dynamic_scale,
                                                                                     int8_mode);
        } else if (residual_num == 2) {
            if (maxbytes >= (48 << 10)) {
                check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    generalAddBiasResidualLayerNorm<T), 2>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes));
            }
            generalAddBiasResidualLayerNorm<T, 2><<<grid, block, maxbytes, stream>>>(input,
                                                                                     residual1,
                                                                                     residual2,
                                                                                     gamma,
                                                                                     beta,
                                                                                     bias,
                                                                                     output,
                                                                                     norm_output,
                                                                                     layernorm_eps,
                                                                                     m,
                                                                                     n,
                                                                                     scale_inter,
                                                                                     scale_out,
                                                                                     scale,
                                                                                     dynamic_scale,
                                                                                     int8_mode);
        }
    }
}

#define INSTANTIATE_INVOKE_GENERAL_ADD_BIAS_RESIDUAL_PRE_LAYER_NORM(T)                \
    template void invokeGeneralAddBiasResidualPreLayerNorm(T *output,                 \
                                                           T *norm_output,            \
                                                           const T *input,            \
                                                           const T *residual1,        \
                                                           const T *residual2,        \
                                                           const T *gamma,            \
                                                           const T *beta,             \
                                                           const T *bias,             \
                                                           const float layernorm_eps, \
                                                           int m,                     \
                                                           int n,                     \
                                                           const float *scale_inter,  \
                                                           const float *scale_out,    \
                                                           float *scale,              \
                                                           float *dynamic_scale,      \
                                                           const int int8_mode,       \
                                                           hipStream_t stream,       \
                                                           int opt_version)
INSTANTIATE_INVOKE_GENERAL_ADD_BIAS_RESIDUAL_PRE_LAYER_NORM(float);
INSTANTIATE_INVOKE_GENERAL_ADD_BIAS_RESIDUAL_PRE_LAYER_NORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_GENERAL_ADD_BIAS_RESIDUAL_PRE_LAYER_NORM(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_GENERAL_ADD_BIAS_RESIDUAL_PRE_LAYER_NORM

} // namespace space_llm
