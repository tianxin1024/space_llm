#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_fp16.h>

#include "kernels/unfused_attention_kernels.h"
#include "kernels/reduce_kernel_utils.cuh"
#include "utils/cuda_utils.h"

namespace space_llm {

__inline__ __device__ int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4) {
    return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template <typename T>
__global__ void addQKVBiasIA3Transpose(T *q_out,
                                       T *k_out,
                                       T *v_out,
                                       const T *__restrict q_in,
                                       const T *__restrict bias_q,
                                       const T *__restrict k_in,
                                       const T *__restrict bias_k,
                                       const T *__restrict v_in,
                                       const T *__restrict bias_v,
                                       const int *ia3_tasks,
                                       const T *ia3_key_weights,
                                       const T *ia3_value_weights,
                                       const int batch_size,
                                       const int seq_len,
                                       const int head_num,
                                       const int size_per_head) {
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id = batch_id * seq_len + word_id;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x) {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id = batch_id * (head_num * seq_len * size_per_head) + head_id * seq_len * size_per_head
                              + word_id * size_per_head + size_id;
        const int src_id = row_id * n + col_id;

        T q = ldg(&q_in[src_id]);
        q_out[target_id] = add(q, ldg(&bias_q[col_id]));

        T k = add(ldg(&k_in[src_id]), ldg(&bias_k[col_id]));
        if (use_ia3_key) {
            k = k * ia3_key_weights[ia3_task * n + col_id];
        }
        k_out[target_id] = k;

        T v = add(ldg(&v_in[src_id]), ldg(&bias_v[col_id]));
        if (use_ia3_value) {
            v = v * ia3_value_weights[ia3_task * n + col_id];
        }
        v_out[target_id] = v;
    }
}

template <typename T>
__global__ void QKVIA3Transpose(T *q_out,
                                T *k_out,
                                T *v_out,
                                const T *__restrict q_in,
                                const T *__restrict k_in,
                                const T *__restrict v_in,
                                const int *ia3_tasks,
                                const T *__restrict ia3_key_weights,
                                const T *__restrict ia3_value_weights,
                                const int batch_size,
                                const int seq_len,
                                const int head_num,
                                const int size_per_head) {
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;                 // batch id
    const int word_id = blockIdx.y;                  // seq_len id
    const int row_id = batch_id * seq_len + word_id; // 行id

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x) {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id = batch_id * (head_num * seq_len * size_per_head) + head_id * seq_len * size_per_head
                              + word_id * size_per_head + size_id;
        const int src_id = row_id * n + col_id;

        q_out[target_id] = ldg(&q_in[src_id]);

        T k = ldg(&k_in[src_id]);
        if (use_ia3_key) {
            k = k * ia3_key_weights[ia3_task * n + col_id];
        }
        k_out[target_id] = k;

        T v = ldg(&v_in[src_id]);
        if (use_ia3_value) {
            v = v * ia3_value_weights[ia3_task * n + col_id];
        }
        v_out[target_id] = v;
    }
}

template <typename T>
void invokeAddQKVBiasIA3Transpose(T *q_buf,
                                  T *k_buf,
                                  T *v_buf,
                                  T *Q,
                                  const T *bias_Q,
                                  T *K,
                                  const T *bias_K,
                                  T *V,
                                  const T *bias_V,
                                  const int batch_size,
                                  const int seq_len,
                                  const int head_num,
                                  const int size_per_head,
                                  const int *ia3_tasks,
                                  const T *ia3_key_weights,
                                  const T *ia3_value_weights,
                                  hipStream_t stream) {
    const int k = head_num * size_per_head;
    dim3 grid(batch_size, seq_len);
    bool is_add_bias = bias_Q != nullptr;

    if (sizeof(T) == 4 || k % 2 != 0) {
        dim3 block(std::min(k, 512));
        if (is_add_bias) {
            addQKVBiasIA3Transpose<T><<<grid, block, 0, stream>>>(q_buf,
                                                                  k_buf,
                                                                  v_buf,
                                                                  Q,
                                                                  bias_Q,
                                                                  K,
                                                                  bias_K,
                                                                  V,
                                                                  bias_V,
                                                                  ia3_tasks,
                                                                  ia3_key_weights,
                                                                  ia3_value_weights,
                                                                  batch_size,
                                                                  seq_len,
                                                                  head_num,
                                                                  size_per_head);
        } else {
            QKVIA3Transpose<T><<<grid, block, 0, stream>>>(q_buf,
                                                           k_buf,
                                                           v_buf,
                                                           Q,
                                                           K,
                                                           V,
                                                           ia3_tasks,
                                                           ia3_key_weights,
                                                           ia3_value_weights,
                                                           batch_size,
                                                           seq_len,
                                                           head_num,
                                                           size_per_head);
        }
        sync_check_cuda_error();
    } else {
        using T2 = typename TypeConverter<T>::Type; // fp16 to half2, bf16 to bf162
        dim3 block(std::min(k / 2, 512));
        if (is_add_bias) {
            addQKVBiasIA3Transpose<T2><<<grid, block, 0, stream>>>((T2 *)q_buf,
                                                                   (T2 *)k_buf,
                                                                   (T2 *)v_buf,
                                                                   (const T2 *)Q,
                                                                   (const T2 *)bias_Q,
                                                                   (const T2 *)K,
                                                                   (const T2 *)bias_K,
                                                                   (const T2 *)V,
                                                                   (const T2 *)bias_V,
                                                                   ia3_tasks,
                                                                   (const T2 *)ia3_key_weights,
                                                                   (const T2 *)ia3_value_weights,
                                                                   batch_size,
                                                                   seq_len,
                                                                   head_num,
                                                                   size_per_head / 2);
        } else {
            QKVIA3Transpose<T2><<<grid, block, 0, stream>>>((T2 *)q_buf,
                                                            (T2 *)k_buf,
                                                            (T2 *)v_buf,
                                                            (const T2 *)Q,
                                                            (const T2 *)K,
                                                            (const T2 *)V,
                                                            ia3_tasks,
                                                            (const T2 *)ia3_key_weights,
                                                            (const T2 *)ia3_value_weights,
                                                            batch_size,
                                                            seq_len,
                                                            head_num,
                                                            size_per_head / 2);
        }
        sync_check_cuda_error();
    }
}

#define INSTANTIATEADDQKVBIASIA3TRANSPOSE(T)                               \
    template void invokeAddQKVBiasIA3Transpose(T *q_buf,                   \
                                               T *k_buf,                   \
                                               T *v_buf,                   \
                                               T *Q,                       \
                                               const T *bias_Q,            \
                                               T *K,                       \
                                               const T *bias_K,            \
                                               T *V,                       \
                                               const T *bias_V,            \
                                               const int batch_size,       \
                                               const int seq_len,          \
                                               const int head_num,         \
                                               const int size_per_head,    \
                                               const int *ia3_tasks,       \
                                               const T *ia3_key_weights,   \
                                               const T *ia3_value_weights, \
                                               hipStream_t stream)
INSTANTIATEADDQKVBIASIA3TRANSPOSE(float);
INSTANTIATEADDQKVBIASIA3TRANSPOSE(half);
#ifdef ENABLE_BF16
INSTANTIATEADDQKVBIASIA3TRANSPOSE(__hip_bfloat16);
#endif
#undef INSTANTIATEADDQKVBIASIA3TRANSPOSE

template <typename T>
__global__ void rebuild_padding_ia3(const T *Q,
                                    const T *K,
                                    const T *V,
                                    T *q_buf_,
                                    T *k_buf_,
                                    T *v_buf_,
                                    const int *ia3_tasks,
                                    const T *ia3_key_weights,
                                    const T *ia3_value_weights,
                                    const int batch_size,
                                    const int seq_len,
                                    const int head_num,
                                    const int size_per_head,
                                    const int *mask_offset) {
    const int bid = blockIdx.x;

    const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
    const int n = head_num * size_per_head;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[tgt_batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        const int tgt_head_id = idx / size_per_head;
        const int tgt_hidden_id = idx % size_per_head;

        const int src_id = bid * n + idx;
        const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + tgt_head_id * seq_len * size_per_head
                           + tgt_seq_id * size_per_head + tgt_hidden_id;

        q_buf_[tgt_id] = ldg(&Q[src_id]);

        T k = ldg(&K[src_id]);
        if (use_ia3_key) {
            k = k * ia3_key_weights[ia3_task * n + idx];
        }
        k_buf_[tgt_id] = k;

        T v = ldg(&V[src_id]);
        if (use_ia3_value) {
            v = v * ia3_value_weights[ia3_task * n + idx];
        }
        v_buf_[tgt_id] = v;
    }
}

template <typename T>
__global__ void add_QKV_bias_rebuild_padding_ia3(const T *Q,
                                                 const T *bias_Q,
                                                 const T *K,
                                                 const T *bias_K,
                                                 const T *V,
                                                 const T *bias_V,
                                                 T *q_buf_,
                                                 T *k_buf_,
                                                 T *v_buf_,
                                                 const int *ia3_tasks,
                                                 const T *ia3_key_weights,
                                                 const T *ia3_value_weights,
                                                 const int batch_size,
                                                 const int seq_len,
                                                 const int head_num,
                                                 const int size_per_head,
                                                 const int *mask_offset) {
    const int bid = blockIdx.x;

    const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
    const int n = head_num * size_per_head;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[tgt_batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        const int tgt_head_id = idx / size_per_head;
        const int tgt_hidden_id = idx % size_per_head;

        const int src_id = bid * n + idx;
        const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + tgt_head_id * seq_len * size_per_head
                           + tgt_seq_id * size_per_head + tgt_hidden_id;

        q_buf_[tgt_id] = add(ldg(&Q[src_id]), ldg(&bias_Q[idx]));

        T k = ldg(&K[src_id]);
        if (use_ia3_key) {
            k = k * ia3_key_weights[ia3_task * n + idx];
        }
        k_buf_[tgt_id] = add(k, ldg(&bias_K[idx]));

        T v = ldg(&V[src_id]);
        if (use_ia3_value) {
            v = v * ia3_value_weights[ia3_task * n + idx];
        }
        v_buf_[tgt_id] = add(v, ldg(&bias_V[idx]));
    }
}

template <typename T>
void invokeAddQKVBiasIA3RebuildPadding(T *Q,
                                       const T *bias_Q,
                                       T *K,
                                       const T *bias_K,
                                       T *V,
                                       const T *bias_V,
                                       T *q_buf,
                                       T *k_buf,
                                       T *v_buf,
                                       const int batch_size,
                                       const int seq_len,
                                       const int head_num,
                                       const int size_per_head,
                                       const int valid_word_num,
                                       const int *mask_offset,
                                       const int *ia3_tasks,
                                       const T *ia3_key_weights,
                                       const T *ia3_value_weights,
                                       hipStream_t stream) {
#ifdef ENABLE_BF16
    bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
#else
    bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
#endif
    using T2 = typename TypeConverter<T>::Type; // fp16 to half2, bf16 to bf162
    int block_size = head_num * size_per_head;
    if (is_half2) {
        while (block_size > 512) {
            if (block_size % 2 == 0) {
                block_size /= 2;
            } else {
                is_half2 = false;
                block_size = std::min(block_size, 512);
                break;
            }
        }
    } else {
        block_size = std::min(block_size, 512);
    }

    if (bias_Q == nullptr && bias_K == nullptr && bias_V == nullptr) {
        if (is_half2) {
            rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>((T2 *)Q,
                                                                           (T2 *)K,
                                                                           (T2 *)V,
                                                                           (T2 *)q_buf,
                                                                           (T2 *)k_buf,
                                                                           (T2 *)v_buf,
                                                                           ia3_tasks,
                                                                           (const T2 *)ia3_key_weights,
                                                                           (const T2 *)ia3_value_weights,
                                                                           batch_size,
                                                                           seq_len,
                                                                           head_num,
                                                                           size_per_head / 2,
                                                                           mask_offset);
        } else {
            rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>(Q,
                                                                           K,
                                                                           V,
                                                                           q_buf,
                                                                           k_buf,
                                                                           v_buf,
                                                                           ia3_tasks,
                                                                           ia3_key_weights,
                                                                           ia3_value_weights,
                                                                           batch_size,
                                                                           seq_len,
                                                                           head_num,
                                                                           size_per_head,
                                                                           mask_offset);
        }
    } else if (bias_Q != nullptr && bias_K != nullptr && bias_V != nullptr) {
        if (is_half2) {
            add_QKV_bias_rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>((T2 *)Q,
                                                                                        (const T2 *)bias_Q,
                                                                                        (T2 *)K,
                                                                                        (const T2 *)bias_K,
                                                                                        (T2 *)V,
                                                                                        (const T2 *)bias_V,
                                                                                        (T2 *)q_buf,
                                                                                        (T2 *)k_buf,
                                                                                        (T2 *)v_buf,
                                                                                        ia3_tasks,
                                                                                        (const T2 *)ia3_key_weights,
                                                                                        (const T2 *)ia3_value_weights,
                                                                                        batch_size,
                                                                                        seq_len,
                                                                                        head_num,
                                                                                        size_per_head / 2,
                                                                                        mask_offset);
        } else {
            add_QKV_bias_rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>(Q,
                                                                                        bias_Q,
                                                                                        K,
                                                                                        bias_K,
                                                                                        V,
                                                                                        bias_V,
                                                                                        q_buf,
                                                                                        k_buf,
                                                                                        v_buf,
                                                                                        ia3_tasks,
                                                                                        ia3_key_weights,
                                                                                        ia3_value_weights,
                                                                                        batch_size,
                                                                                        seq_len,
                                                                                        head_num,
                                                                                        size_per_head,
                                                                                        mask_offset);
        }
    } else {
        QK_CHECK(false);
    }
}

#define INSTANTIATEADDQKVBIASIA3REBUILDPADDING(T)                               \
    template void invokeAddQKVBiasIA3RebuildPadding(T *Q,                       \
                                                    const T *bias_Q,            \
                                                    T *K,                       \
                                                    const T *bias_K,            \
                                                    T *V,                       \
                                                    const T *bias_V,            \
                                                    T *q_buf,                   \
                                                    T *k_buf,                   \
                                                    T *v_buf,                   \
                                                    const int batch_size,       \
                                                    const int seq_len,          \
                                                    const int head_num,         \
                                                    const int size_per_head,    \
                                                    const int valid_word_num,   \
                                                    const int *mask_offset,     \
                                                    const int *ia3_tasks,       \
                                                    const T *ia3_key_weights,   \
                                                    const T *ia3_value_weights, \
                                                    hipStream_t stream)
INSTANTIATEADDQKVBIASIA3REBUILDPADDING(float);
INSTANTIATEADDQKVBIASIA3REBUILDPADDING(half);
#ifdef ENABLE_BF16
INSTANTIATEADDQKVBIASIA3REBUILDPADDING(__hip_bfloat16);
#endif
#undef INSTANTIATEADDQKVBIASIA3REBUILDPADDING

template <typename T>
__global__ void addRelativeAttentionBias(
    T *qk_buf, const T *relative_attention_bias, const int batch_size, const int head_num, const int seq_len) {
    for (int i = threadIdx.x; i < batch_size * seq_len; i += blockDim.x) {
        int batch_id = i / seq_len;
        int seq_id = i % seq_len;

        const int bias_index = blockIdx.x * seq_len + seq_id;
        const int qk_index = batch_id * gridDim.x * seq_len + bias_index;
        qk_buf[qk_index] = add(qk_buf[qk_index], relative_attention_bias[bias_index]);
    }
}

template <typename T>
void invokeAddRelativeAttentionBias(T *qk_buf,
                                    const T *relative_attention_bias,
                                    const int batch_size,
                                    const int head_num,
                                    const int seq_len,
                                    hipStream_t stream) {
    // qk_buf: [batch_size, head_num, seq_len, seq_len]
    // relative_attention_bias: [1, head_num, seq_len, seq_len]
    dim3 grid(head_num * seq_len);
    dim3 block(512);
    using T2 = typename TypeConverter<T>::Type;

#ifdef ENABLE_BF16
    const bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (seq_len % 2 == 0);
#else
    const bool is_half2 = (std::is_same<T, half>::value) && (seq_len % 2 == 0);
#endif
    if (is_half2) {
        addRelativeAttentionBias<T2><<<grid, block, 0, stream>>>(
            (T2 *)qk_buf, (const T2 *)relative_attention_bias, batch_size, head_num, seq_len / 2);
    } else {
        addRelativeAttentionBias<<<grid, block, 0, stream>>>(
            qk_buf, relative_attention_bias, batch_size, head_num, seq_len);
    }
}

#define INSTANTIATEADDRELATIVEATTENTIONBIAS(T)                                     \
    template void invokeAddRelativeAttentionBias(T *qk_buf,                        \
                                                 const T *relative_attention_bias, \
                                                 const int batch_size,             \
                                                 const int head_num,               \
                                                 const int seq_len,                \
                                                 hipStream_t stream)

INSTANTIATEADDRELATIVEATTENTIONBIAS(float);
INSTANTIATEADDRELATIVEATTENTIONBIAS(half);
#ifdef ENABLE_BF16
INSTANTIATEADDRELATIVEATTENTIONBIAS(__hip_bfloat16);
#endif
#undef INSTANTIATEADDRELATIVEATTENTIONBIAS

template <typename T, typename T_IN, int ITEMS_PER_THREAD>
__global__ void softmax_kernel(T *attn_score,
                               const T_IN *qk,
                               const T *attn_mask,
                               const T *linear_bias_slopes,
                               const int batch_size,
                               const int head_num,
                               const int q_length,
                               const int k_length,
                               const float qk_scale) {
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]
    const int64_t bi = blockIdx.y; // Batch index.
    const int64_t hi = blockIdx.z; // Head index.

    __shared__ float s_mean, s_max;

    const float linear_bias_slope = linear_bias_slopes != nullptr ? (float)linear_bias_slopes[hi] : 0.0f;

    for (int64_t qi = blockIdx.x; qi < q_length; qi += gridDim.x) {
        float data[ITEMS_PER_THREAD];
        int64_t qk_offset;
        float local_max = -1e20f;
        // Loop along with K dimension.
        for (int64_t i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            int64_t ki = blockDim.x * i + threadIdx.x; // Index of K dimension.
            qk_offset = ((bi * head_num + hi) * q_length + qi) * k_length + ki;

            float qk_val = static_cast<float>(qk[qk_offset]);
            float qk_bias = 0.0f;

            if (linear_bias_slopes != nullptr) {
                // We don't handle the upper diagonal (ki > qi) separately, whose values
                // are negligible due to the negative infinity mask. And it matches with
                // the HF's implementation.
                qk_bias += static_cast<float>(linear_bias_slope * (ki - qi));
            }

            int64_t mask_offset = (bi * q_length + qi) * k_length + ki;
            float mask_val = static_cast<float>(ldg(&attn_mask[mask_offset]));
            qk_bias += (1.0f - mask_val) * -10000.0f;

            data[i] = qk_scale * qk_val + qk_bias;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int64_t i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int64_t i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            qk_offset = ((bi * head_num + hi) * q_length + qi) * k_length + blockDim.x * i + threadIdx.x;
            attn_score[qk_offset] = (T)(data[i] * s_mean);
        }
    }
}

template <typename T, int K_ITEMS_PER_THREAD, int Q_ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2_v2(T *attn_score,
                                     const T *qk_buf,
                                     const T *attn_mask,
                                     const T *linear_bias_slopes,
                                     const int batch_size,
                                     const int head_num,
                                     const int q_length,
                                     const int k_length,
                                     const T scalar) {
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    // QK^T matrix of shape (batch_size, head_num, q_length, k_length / 2)
    T2 *attn_score_h2 = reinterpret_cast<T2 *>(attn_score);
    const T2 *qk_buf_h2 = reinterpret_cast<const T2 *>(qk_buf);
    const T2 *attn_mask_h2 = reinterpret_cast<const T2 *>(attn_mask);

    const int bi = blockIdx.y; // Batch index
    const int hi = blockIdx.z; // Head index.

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE = cuda_cast<T2>(1.0f);
    const T2 ZERO = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale = cuda_cast<T2>(scalar);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    __shared__ float s_sum[Q_ITEMS_PER_THREAD], s_max[Q_ITEMS_PER_THREAD];

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x * Q_ITEMS_PER_THREAD) {
        T2 data[Q_ITEMS_PER_THREAD][K_ITEMS_PER_THREAD];

        int qk_offset[Q_ITEMS_PER_THREAD];

        float local_max[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_max[j] = -1e20f;
        }

        // Loop over k dimension.
        const int Q_ITEMS = min((q_length - qi + gridDim.x - 1) / gridDim.x, Q_ITEMS_PER_THREAD);
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki = blockDim.x * i + threadIdx.x;

            int mask_offset[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
                mask_offset[j] = (bi * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
            }

            T2 mask_val[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = ldg(&attn_mask_h2[mask_offset[j]]);
            }

            T2 qk[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk[j] = qk_buf_h2[qk_offset[j]];
            }

            T2 pos_bias[Q_ITEMS_PER_THREAD];
            if (linear_bias_slopes != nullptr) {
#pragma unroll
                for (int j = 0; j < Q_ITEMS; j++) {
                    // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                    // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                    // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                    // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                    // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                    // separately, whose values are negligible due to the negative infinity mask.
                    int qidx = qi + j * gridDim.x;
                    T2 dist(2.0f * ki - qidx, 2.0f * ki + 1 - qidx);
                    pos_bias[j] = hmul2<T2>(linear_bias_slope, dist);
                }
            }
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = hmul2<T2>(hsub2<T2>(ONE, mask_val[j]), NEG_INFTY);
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                T2 val = hadd2<T2>(hmul2<T2>(qk_scale, qk[j]), mask_val[j]);
                if (linear_bias_slopes != nullptr) {
                    val = hadd2<T2>(val, pos_bias[j]);
                }
                data[j][i] = val;
                local_max[j] = fmax(local_max[j], fmax((float)data[j][i].x, (float)data[j][i].y));
            }
        }

        if (blockDim.x <= 32) {
            warpReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        } else {
            blockReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_sum[j] = {0.f};
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; ++j) {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], cuda_cast<T2>(s_max[j])));
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                local_sum[j] += (float)(data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32) {
            warpReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        } else {
            blockReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + blockDim.x * i
                               + threadIdx.x;
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                attn_score_h2[qk_offset[j]] = hmul2<T2>(data[j][i], cuda_cast<T2>(s_sum[j]));
            }
        }
    }
}

template <typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2(T *attn_score,
                                  const T *qk_buf,
                                  const T *attn_mask,
                                  const T *linear_bias_slopes,
                                  const int batch_size,
                                  const int head_num,
                                  const int q_length,
                                  const int k_length,
                                  const T qk_scale) {
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type; // half2

    T2 *attn_score_h2 = reinterpret_cast<T2 *>(attn_score);
    const T2 *qk_buf_h2 = reinterpret_cast<const T2 *>(qk_buf);
    const T2 *attn_mask_h2 = reinterpret_cast<const T2 *>(attn_mask);

    const int bi = blockIdx.y; // Batch index
    const int hi = blockIdx.z; // Head index

    __shared__ float s_mean, s_max;

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE = cuda_cast<T2>(1.0f);
    const T2 ZERO = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale_h2 = cuda_cast<T2>(qk_scale);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    // Loop over Q dimention.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x) {
        T2 data[ITEMS_PER_THREAD];
        int qk_offset;
        float local_max = -1e20f;

        // Loop over K dimention
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            // The half of the index of k dimention. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki = blockDim.x * i + threadIdx.x;
            qk_offset = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + ki;
            int mask_offset = (bi * q_length + qi) * (k_length / 2) + ki;

            // The value of QK^T matrix at (qi, ki).
            T2 qk = qk_buf_h2[qk_offset];
            // The bias value to the position (qi, ki) including both mask and positional bias.
            T2 qk_bias = ZERO;

            if (linear_bias_slopes != nullptr) {
                // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                // separately, whose values are negligible due to the negative infinity mask.
                T2 dist(2.0f * ki - qi, 2.0f * ki + 1 - qi);
                qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(linear_bias_slope, dist));
            }

            T2 mask_val = ldg(&attn_mask_h2[mask_offset]);
            qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(hsub2<T2>(ONE, mask_val), NEG_INFTY));

            data[i] = hadd2<T2>(hmul2<T2>(qk, qk_scale_h2), qk_bias);
            local_max = fmax(local_max, fmax((float)data[i].x, (float)data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0.0f;
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], cuda_cast<T2>(s_max)));
            local_sum += (float)(data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + blockDim.x * i + threadIdx.x;
            attn_score_h2[qk_offset] = hmul2<T2>(data[i], cuda_cast<T2>(s_mean));
        }
    }
}

#define LAUNCH_MAKSED_SOFTMAX_(T_, ITEMS_PER_THREAD)                                                                  \
    block.x /= ITEMS_PER_THREAD;                                                                                      \
    block.x = (block.x + 31) / 32 * 32;                                                                               \
    assert(block.x <= 1024);                                                                                          \
    if (is_half2) {                                                                                                   \
        if (grid.x % 4 == 0) {                                                                                        \
            grid.x /= 4;                                                                                              \
            softmax_kernel_h2_v2<T_, ITEMS_PER_THREAD, 4>                                                             \
                <<<grid, block, 0, stream>>>((T_ *)param.attention_score,                                             \
                                             (const T_ *)param.qk,                                                    \
                                             (const T_ *)param.attention_mask,                                        \
                                             (const T_ *)param.linear_bias_slopes,                                    \
                                             param.batch_size,                                                        \
                                             param.num_heads,                                                         \
                                             param.q_length,                                                          \
                                             param.k_length,                                                          \
                                             (const T_)param.qk_scale);                                               \
        } else {                                                                                                      \
            softmax_kernel_h2<T_, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>((T_ *)param.attention_score,          \
                                                                                (const T_ *)param.qk,                 \
                                                                                (const T_ *)param.attention_mask,     \
                                                                                (const T_ *)param.linear_bias_slopes, \
                                                                                param.batch_size,                     \
                                                                                param.num_heads,                      \
                                                                                param.q_length,                       \
                                                                                param.k_length,                       \
                                                                                (const T_)param.qk_scale);            \
        }                                                                                                             \
    } else {                                                                                                          \
        softmax_kernel<T, T_IN, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>(param.attention_score,                  \
                                                                              param.qk,                               \
                                                                              param.attention_mask,                   \
                                                                              param.linear_bias_slopes,               \
                                                                              param.batch_size,                       \
                                                                              param.num_heads,                        \
                                                                              param.q_length,                         \
                                                                              param.k_length,                         \
                                                                              param.qk_scale);                        \
    }

#define LAUNCH_MAKSED_SOFTMAX(ITEMS_PER_THREAD) LAUNCH_MAKSED_SOFTMAX_(half, ITEMS_PER_THREAD)

template <typename T, typename T_IN>
void invokeMaskedSoftmax(MaskedSoftmaxParam<T, T_IN> &param, hipStream_t stream) {
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX(4)
    } else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX(2)
    } else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX(1)
    } else {
        QK_CHECK(param.k_length <= 4096);
    }
}

template void invokeMaskedSoftmax(MaskedSoftmaxParam<float, float> &param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, float> &param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, half> &param, hipStream_t stream);

template <typename T>
__global__ void transpose_attentions(
    T *attentions_out, const T *attentions_in, size_t batch_size, size_t num_layers, size_t num_heads, size_t seq_len) {
    // attentions_in  shape [B, H, S, S]
    // attentions_out shape [B, L, H, S, S].
    // Note that we write the L dimension as if it was index 0.
    // In reality, the pointer has already been shifted to point to the correct layer.

    const auto batch_idx = blockIdx.x;
    const auto head_idx = blockIdx.y;

    const auto dst_offset = (batch_idx * num_layers * num_heads + head_idx) * seq_len * seq_len;
    const auto src_offset = (batch_idx * num_heads + head_idx) * seq_len * seq_len;

    for (auto x = threadIdx.x; x < seq_len * seq_len; x += blockDim.x) {
        attentions_out[dst_offset + x] = attentions_in[src_offset + x];
    }
}

template <typename T>
void invokeTransposeAttentions(Tensor &attentions_out, const Tensor &attentions_in, hipStream_t stream) {
    const size_t batch_size = attentions_in.shape[0];
    const size_t num_heads = attentions_in.shape[1];
    const size_t seq_len = attentions_in.shape[2];
    const size_t num_layers = attentions_out.shape[1];

    const dim3 gridSize(batch_size, num_heads);
    const dim3 blockSize(512);

    transpose_attentions<<<gridSize, blockSize, 0, stream>>>(
        attentions_out.getPtr<T>(), attentions_in.getPtr<const T>(), batch_size, num_layers, num_heads, seq_len);
}

#define INSTANTIATETRANSPOSEATTENTIONS(T)       \
    template void invokeTransposeAttentions<T>( \
        Tensor & attentions_out, const Tensor &attentions_in, hipStream_t stream)
INSTANTIATETRANSPOSEATTENTIONS(float);
INSTANTIATETRANSPOSEATTENTIONS(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEATTENTIONS(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEATTENTIONS

template <typename T>
__global__ void transpose(const T *src,
                          T *dst,
                          const int batch_size,
                          const int seq_len,
                          const int head_num,
                          const int size_per_head,
                          const float *scale,
                          int int8_mode) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int batch_id = tid / (head_num * seq_len * size_per_head);
    int head_id = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
    int seq_id = (tid % (seq_len * size_per_head)) / size_per_head;
    int id = tid % size_per_head;

    int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);

    if (int8_mode == 2) {
        using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
        using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;

        const Float_Packed_T scale_val = cuda_cast<Float_Packed_T>(*scale);
        reinterpret_cast<Int8_Packed_T *>(dst)[target_id] =
            cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src[tid]) * scale_val);
    } else {
        dst[target_id] = src[tid];
    }
}

template <>
__global__ void transpose(const float *src,
                          float *dst,
                          const int batch_size,
                          const int seq_len,
                          const int head_num,
                          const int size_per_head,
                          const float *scale,
                          int int8_mode) {
    int batch_id = blockIdx.x / (head_num * seq_len);
    int seq_id = blockIdx.x % seq_len;
    int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;

    const int target_id = batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
                          + head_id * size_per_head + threadIdx.x;
    const int src_id = blockIdx.x * size_per_head + threadIdx.x;

    if (int8_mode == 2) {
        const float scale_val = *scale;
        reinterpret_cast<int8_t *>(dst)[target_id] = cuda_cast<int8_t>(src[src_id] * scale_val);
    } else {
        dst[target_id] = src[src_id];
    }
}

template <typename T>
void invokeTransposeQKV(T *dst,
                        T *src,
                        const int batch_size,
                        const int seq_len,
                        const int head_num,
                        const int size_per_head,
                        const float *scale,
                        const int int8_mode,
                        hipStream_t stream) {
    dim3 grid, block;
    if (sizeof(T) == 2) {
        int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        while (seq_per_block < 4 && grid.x % 2 == 0) {
            grid.x /= 2;
            seq_per_block *= 2;
        }

        QK_CHECK(grid.x * seq_per_block == (size_t)batch_size * head_num * seq_len);

        if (seq_per_block * size_per_head % 2 == 0) {
            block.x = seq_per_block * size_per_head / 2;
            if (std::is_same<T, half>::value) {
                transpose<half2><<<grid, block, 0, stream>>>(
                    (half2 *)src, (half2 *)dst, batch_size, seq_len, head_num, size_per_head / 2, scale, int8_mode);
            }
#ifdef ENABLE_BF16
            else {
                transpose<__hip_bfloat162><<<grid, block, 0, stream>>>((__hip_bfloat162 *)src,
                                                                      (__hip_bfloat162 *)dst,
                                                                      batch_size,
                                                                      seq_len,
                                                                      head_num,
                                                                      size_per_head / 2,
                                                                      scale,
                                                                      int8_mode);
            }
#endif
        } else {
            block.x = seq_per_block * size_per_head;
            transpose<T>
                <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
        }
    } else {
        const int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        block.x = seq_per_block * size_per_head;
        transpose<T>
            <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
    }
}

#define INSTANTIATETRANSPOSEQKV(T)                            \
    template void invokeTransposeQKV(T *src,                  \
                                     T *dst,                  \
                                     const int batch_size,    \
                                     const int seq_len,       \
                                     const int head_num,      \
                                     const int size_per_head, \
                                     const float *scale,      \
                                     const int int8_mode,     \
                                     hipStream_t stream)
INSTANTIATETRANSPOSEQKV(float);
INSTANTIATETRANSPOSEQKV(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEQKV(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEQKV

template <typename T>
__global__ void transpose_remove_padding(const T *src,
                                         T *dst,
                                         const int batch_size,
                                         const int seq_len,
                                         const int head_num,
                                         const int size_per_head,
                                         const int *mask_offset,
                                         const float *scale,
                                         const int int8_mode) {
    // TODO: optimize this kernel?
    // do remove_sequence_length_padding
    const int bid = blockIdx.x; // batch * seq_len or valid_word_num

    const int src_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int src_seq_id = (bid + mask_offset[bid]) % seq_len;

    const int dst_seq_id = bid;

    const int src_offset_base = src_batch_id * seq_len * head_num * size_per_head + src_seq_id * size_per_head;
    const int dst_offset_base = dst_seq_id * head_num * size_per_head;

    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    const Float_Packed_T scale_val =
        int8_mode == 2 ? cuda_cast<Float_Packed_T>(*scale) : cuda_cast<Float_Packed_T>(0.0f);

    for (int idx = threadIdx.x; idx < head_num * size_per_head; idx += blockDim.x) {
        const int head_id = idx / size_per_head;
        const int hidden_id = idx % size_per_head;
        const T src_elem = ldg(&src[src_offset_base + head_id * seq_len * size_per_head + hidden_id]);
        if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T *>(dst)[dst_offset_base + idx] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src_elem) * scale_val);
        } else {
            dst[dst_offset_base + idx] = src_elem;
        }
    }
}

// clang-format off
template<typename T>
void invokeTransposeAttentionOutRemovePadding(T*           src,
                                              T*           dst,
                                              const int    valid_word_num,
                                              const int    batch_size,
                                              const int    seq_len,
                                              const int    head_num,
                                              const int    size_per_head,
                                              const int*   mask_offset,
                                              const float* scale,
                                              const int    int8_mode,
                                              hipStream_t stream)
{
#ifdef ENABLE_BF16
    bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
#else
    bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
#endif
    using T2       = typename TypeConverter<T>::Type;  // fp16 to half2, bf16 to bf162
    int block_size = head_num * size_per_head;
    if (is_half2) {
        while (block_size > 512) {
            if (block_size % 2 == 0) {
                block_size /= 2;
            }
            else {
                is_half2   = false;
                block_size = std::min(block_size, 1024);
                break;
            }
        }
    }
    else {
        block_size = std::min(block_size, 1024);
    }

    if (is_half2) {
        transpose_remove_padding<T2><<<valid_word_num, block_size, 0, stream>>>(
            (T2*)src, (T2*)dst, batch_size, seq_len, head_num, size_per_head / 2, mask_offset, scale, int8_mode);
    }
    else {
        transpose_remove_padding<<<valid_word_num, block_size, 0, stream>>>(
            src, dst, batch_size, seq_len, head_num, size_per_head, mask_offset, scale, int8_mode);
    }
}
// clang-format on

#define INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(T)                             \
    template void invokeTransposeAttentionOutRemovePadding(T *src,                   \
                                                           T *dst,                   \
                                                           const int valid_word_num, \
                                                           const int batch_size,     \
                                                           const int seq_len,        \
                                                           const int head_num,       \
                                                           const int size_per_head,  \
                                                           const int *mask_offset,   \
                                                           const float *scale,       \
                                                           const int int8_mode,      \
                                                           hipStream_t stream)
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(float);
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING

} // namespace space_llm
